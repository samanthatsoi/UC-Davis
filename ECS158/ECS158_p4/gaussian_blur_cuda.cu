
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <string.h>
#include <stdio.h>
#include <time.h>
#include <math.h>


#define DIV_ROUND_UP(n, d)  (((n) + (d) - 1) / (d))
#define cuda_check(ret) _cuda_check((ret), __FILE__, __LINE__)
inline void _cuda_check(hipError_t ret, const char *file, int line)
{
  if (ret != hipSuccess) {
    fprintf(stderr, "CudaErr: %s (%s:%d)\n", hipGetErrorString(ret), file, line);
    exit(1);
  }
}
void print_matrix(float* matrix, int order){
  for (int row=0; row<order; row++)
  {
      for(int columns=0; columns<order; columns++)
          {
           printf("%f ", matrix[row*order + columns]);
          }
      printf("\n");
   }
}
float Gaussian(int x, int y, float sigma){
  return (1.0 / (2.0 * M_PI * sigma * sigma)) * exp(-((x*x+y*y)/(2*sigma*sigma)));
}

__global__ void calculation_kernel(float *Gaussian_matrix, unsigned char* image_buffer, unsigned char*output_matrix, int order, int width, int height)
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
      float sum = 0;
      //traversing the Gaussian_matrix
      for(unsigned int k = 0; k < order; k++)
      {
        for(unsigned int z = 0; z < order; z++)
        {
          int locX = (col - (floor((float)order / 2))) + z;
          int locY = (row - (floor((float)order / 2))) + k;
          // Getting nearest pixel
          if(locX < 0)
          {
            locX = 0;
          }
          else if(locX >= width)
          {
            locX = width-1;
          }

          if(locY < 0)
          {
            locY = 0;
          }
          else if(locY >= height)
          {
            locY = height-1;
          }
          float temp =  Gaussian_matrix[k*order+z] *image_buffer[(locY * width  + locX)];
          sum += temp;
        }
      }
  output_matrix[row*width+col] = (unsigned char)sum;
}

// Send every Pixel to GPU for computation
void calculation(float *Gaussian_matrix_h, unsigned char* image_buffer_h, unsigned char * output_matrix_h, int order, int width, int height)
{
  float * Gaussian_matrix_d;
  unsigned char *image_buffer_d;
  unsigned char *output_matrix_d;
  int kernel_size = order * order * sizeof(float);
  int image_size = width * height * sizeof(unsigned char);

  cuda_check(hipMalloc(&Gaussian_matrix_d, kernel_size));
  cuda_check(hipMalloc(&image_buffer_d, image_size));
  cuda_check(hipMalloc(&output_matrix_d, image_size));
  cuda_check(hipMemcpy(Gaussian_matrix_d, Gaussian_matrix_h, kernel_size, hipMemcpyHostToDevice));
  cuda_check(hipMemcpy(image_buffer_d, image_buffer_h, image_size, hipMemcpyHostToDevice));
  dim3 block_dim(32,32);
  dim3 grid_dim(DIV_ROUND_UP(width, block_dim.x), DIV_ROUND_UP(height, block_dim.y));
  calculation_kernel<<<grid_dim, block_dim>>>(Gaussian_matrix_d, image_buffer_d, output_matrix_d, order, width, height);
  cuda_check(hipPeekAtLastError());      /* Catch configuration errors */
  cuda_check(hipDeviceSynchronize());    /* Catch execution errors */
  cuda_check(hipMemcpy(output_matrix_h, output_matrix_d, image_size, hipMemcpyDeviceToHost));
  cuda_check(hipFree(Gaussian_matrix_d));
  cuda_check(hipFree(image_buffer_d));
  cuda_check(hipFree(output_matrix_d));
}

int main(int argc, char *argv[]){
  struct timespec start, end;
  if(argc != 4) {
    fprintf(stderr, "Usage: ./gaussian_blur_cuda <input_file> <output_file> <sigma>");
    return 1;
  }
  float sigma = atof(argv[3]);
  FILE *fp;
  fp = fopen(argv[1], "rb");
  char buffer[4];
  unsigned  int width;
  unsigned int height;
  int cutoff;

  if (!fp) {
    fprintf(stderr, "Error: cannot open file %s", argv[1]);
    return 1;
  }
  if(fscanf(fp,"%s\n",buffer) != 1){
    fprintf(stderr, "Error: invalid PGM information");
    return 1;
  }
  if(strcmp(buffer,"P5") != 0){
    fprintf(stderr, "Error: invalid PGM information");
    return 1;
  }
  if( fscanf(fp, "%d %d\n", &width, &height) != 2){         //taking in two values
    fprintf(stderr, "Error: invalid PGM information");
    return 1;
  }
  if(width < 256 || height < 256){
    fprintf(stderr, "Error: invalid PGM pixels");
    return 1;
  }
  if( fscanf(fp, "%d\n", &cutoff)!= 1){
    fprintf(stderr, "Error: invalid PGM information");
    return 1;
  }
  if(cutoff != 255){
    fprintf(stderr, "Error: invalid PGM information");
    return 1;
  }
  if(sigma == 0){
    fprintf(stderr, "Error: invalid sigma value");
    return 1;
  }
  unsigned  int order = ceil(6*sigma);
  if(order % 2 == 0){
    order++;
  }
  if(order > height ||order  > width){
     fprintf(stderr, "Error: sigma value too big for image size");
     return 1;
  }

  float *Gaussian_matrix;
  unsigned char *image_buffer;
  unsigned char *output_matrix;
  Gaussian_matrix = (float*)malloc(order * order * sizeof(float));
  image_buffer = (unsigned char*)malloc(width * height * sizeof(unsigned char));
  output_matrix = (unsigned char*)malloc(width * height * sizeof(unsigned char));
  for(unsigned int x = 0, i = 0; x <= 2*floor(order/2) && i < order; x++, i++){
    for(unsigned int y = 0, j = 0; y <= 2*floor(order/2) && j < order; y++, j++){
      Gaussian_matrix[i*order+j] = Gaussian(x-floor(order/2),y-floor(order/2),sigma);
    }
  }
  if(fread(image_buffer, sizeof(unsigned char), height * width, fp) != width*height){
    fprintf(stderr, "Error: invalid PGM pixels");
    return 1;
  }
  clock_gettime(CLOCK_MONOTONIC, &start);
  calculation(Gaussian_matrix, image_buffer, output_matrix, order, width, height);
  clock_gettime(CLOCK_MONOTONIC, &end);
  double diff = end.tv_sec - start.tv_sec + (end.tv_nsec - start.tv_nsec)/1000000000.0;
  printf("Running time: %f secs\n", diff);
  FILE* pgmimg;
  char filename[40];
  strcpy(filename, argv[2]);
  pgmimg = fopen( filename , "wb" );
  if(pgmimg == NULL){
    fprintf(stderr, "cannot open file to write");
    return 1;
  }
  fprintf(pgmimg, "P5\n");
  fprintf(pgmimg, "%d %d\n", width, height);
  fprintf(pgmimg, "%d\n", cutoff);
  fwrite(output_matrix, sizeof(unsigned char), height * width, pgmimg);
  fclose(pgmimg);
  fclose(fp);
  return 0;
}
